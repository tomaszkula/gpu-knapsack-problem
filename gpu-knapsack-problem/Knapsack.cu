#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

__global__ void fillArray(int* _array, int n, int _value)
{
    int _id = threadIdx.x + blockIdx.x * blockDim.x;
    if (_id < n)
    {
        _array[_id] = _value;
    }
}

__device__ int maxi(int _a, int _b)
{
    if (_a > _b)
    {
        return _a;
    }
    else
    {
        return _b;
    }
}

__global__ void knapsackKernel(int i, int _xSize, int *_weights, int *_profits, int *maxValues)
{
    int j = threadIdx.x;

    int _id1 = (i * _xSize) + j;
    int _id2 = ((i - 1) * _xSize) + j;

    if (i == 0 || j == 0)
    {
        maxValues[_id1] = 0;
    }
    else if (_weights[i - 1] <= j)
    {
        maxValues[_id1] = maxi(maxValues[((i - 1) * _xSize) + (j - _weights[i - 1])] + _profits[i - 1], maxValues[_id2]);
    }
    else
    {
        maxValues[_id1] = maxValues[_id2];
    }

    __syncthreads();
}

extern "C"
{
    __host__ void solveParallelly(int _capacity, std::vector<int> _weights, std::vector<int> _profits)
    {
        int _intSize = sizeof(int);
        int _weightCount = _weights.size();
        int _weightsSize = _weightCount * _intSize;
        int _profitsCount = _profits.size();
        int _profitsSize = _profitsCount * _intSize;

        int* _dCapacity, * _dWeights, * _dProfits;
        hipMalloc((void**)&_dCapacity, _intSize);
        hipMalloc((void**)&_dWeights, _weightsSize);
        hipMalloc((void**)&_dProfits, _profitsSize);
        hipMemcpy(_dCapacity, &_capacity, _intSize, hipMemcpyHostToDevice);
        hipMemcpy(_dWeights, &_weights[0], _weightsSize, hipMemcpyHostToDevice);
        hipMemcpy(_dProfits, &_profits[0], _profitsSize, hipMemcpyHostToDevice);

        int _maxValuesCount = (_weightCount + 1) * (_capacity + 1);
        int _maxValuesSize = _maxValuesCount * _intSize;
        int* _maxValues = (int*)malloc(_maxValuesSize);
        int* _dMaxValues; hipMalloc((void**)&_dMaxValues, _maxValuesSize);
        //fillArray <<<_maxValuesCount, 1 >>> (_dMaxValues, _maxValuesCount, 0);
        //hipMemcpy(_maxValues, _dMaxValues, _maxValuesSize, hipMemcpyDeviceToHost);

        for (int i = 0; i <= _weightCount; i++)
        {
            knapsackKernel << <1, _capacity + 1 >> > (i, _capacity + 1, _dWeights, _dProfits, _dMaxValues);
        }

        hipDeviceSynchronize();
        hipMemcpy(_maxValues, _dMaxValues, _maxValuesSize, hipMemcpyDeviceToHost);

        std::cout << "BEST VALUE : " << _maxValues[_weightCount * (_capacity + 1) + _capacity] << std::endl;

        free(_maxValues);

        hipFree(_dCapacity);
        hipFree(_dWeights);
        hipFree(_dProfits);
        hipFree(_dMaxValues);
    }

    __host__ float testParallelly(int _capacity, std::vector<int> _weights, std::vector<int> _profits, int _times)
    {
        hipEvent_t _startEvent, _stopEvent;
        hipEventCreate(&_startEvent);
        hipEventCreate(&_stopEvent);

        int _intSize = sizeof(int);
        int _weightCount = _weights.size();
        int _weightsSize = _weightCount * _intSize;
        int _profitsCount = _profits.size();
        int _profitsSize = _profitsCount * _intSize;

        int* _dCapacity, * _dWeights, * _dProfits;
        hipMalloc((void**)&_dCapacity, _intSize);
        hipMalloc((void**)&_dWeights, _weightsSize);
        hipMalloc((void**)&_dProfits, _profitsSize);
        hipMemcpy(_dCapacity, &_capacity, _intSize, hipMemcpyHostToDevice);
        hipMemcpy(_dWeights, &_weights[0], _weightsSize, hipMemcpyHostToDevice);
        hipMemcpy(_dProfits, &_profits[0], _profitsSize, hipMemcpyHostToDevice);

        int _maxValuesCount = (_weightCount + 1) * (_capacity + 1);
        int _maxValuesSize = _maxValuesCount * _intSize;
        int* _maxValues = (int*)malloc(_maxValuesSize);
        int* _dMaxValues; hipMalloc((void**)&_dMaxValues, _maxValuesSize);

        hipEventRecord(_startEvent);
        for (int t = 0; t < _times; t++)
        {
            for (int i = 0; i <= _weightCount; i++)
            {
                knapsackKernel << <1, _capacity + 1 >> > (i, _capacity + 1, _dWeights, _dProfits, _dMaxValues);
            }
        }
        hipEventRecord(_stopEvent);

        hipDeviceSynchronize();
        hipMemcpy(_maxValues, _dMaxValues, _maxValuesSize, hipMemcpyDeviceToHost);

        hipEventSynchronize(_stopEvent);
        float _duration = 0;
        hipEventElapsedTime(&_duration, _startEvent, _stopEvent);

        free(_maxValues);

        hipFree(_dCapacity);
        hipFree(_dWeights);
        hipFree(_dProfits);
        hipFree(_dMaxValues);

        return _duration;
    }
}